#include "hip/hip_runtime.h"
#include "cudaKernels.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>

//------------------------------------------------------------------------------
// A do-nothing kernel
//------------------------------------------------------------------------------
__global__ void noOpKernel()
{
    // No operation
}

//------------------------------------------------------------------------------
// Map resource, then unmap, do nothing
//------------------------------------------------------------------------------
void ApplyCudaKernel(hipGraphicsResource* cudaResource, int width, int height)
{
    if (!cudaResource) return;

    // Map the D3D11 resource into CUDA
    hipError_t cuErr = hipGraphicsMapResources(1, &cudaResource, 0);
    if (cuErr != hipSuccess) {
        printf("hipGraphicsMapResources failed: %s\n", hipGetErrorString(cuErr));
        return;
    }

    // Optionally run noOpKernel for demonstration:
    noOpKernel << <1, 1 >> > ();
    hipDeviceSynchronize();

    // Unmap so D3D can use it again
    hipGraphicsUnmapResources(1, &cudaResource, 0);
}
